#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <regex>

#include <hipcub/hipcub.hpp>

#include "../cuda.cuh"

const size_t blockSize = 32;
const size_t warp_size = 32;

std::vector<char> prepare_array(std::vector<std::string>& passwords) {
    std::vector<char> char_array(warp_size * passwords.size()); // No pw longer than 32 (warp size)
    for (size_t password_idx = 0; password_idx < passwords.size(); password_idx++) {
        size_t char_idx;
        for (char_idx = 0; char_idx < passwords[password_idx].size(); char_idx++) {
            char_array[password_idx + char_idx * passwords.size()] = passwords[password_idx][char_idx];
        }
        for (size_t pad_idx = char_idx; pad_idx < warp_size; pad_idx++) {
            char_array[password_idx + pad_idx * passwords.size()] = '0';
        }
    }
    return char_array;
}

template <typename T>
void array_to_device(T*& array_ptr, std::vector<T>& data) {
    CUDA_CALL(hipMalloc((void** )&array_ptr, data.size() * sizeof(T)));
    CUDA_CALL(hipMemcpy(array_ptr, data.data(), data.size() * sizeof(T),
                         hipMemcpyDefault));
}

__global__
void invalidate_passwords(char* passwords, char* policy,
                          int* lower, int* upper, char* valid,
                          size_t input_len) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < input_len) {
        char policy_char = policy[index];
        if (!((passwords[index + (upper[index] - 1) * input_len] == policy_char) ^
             passwords[index + (lower[index] - 1) * input_len] == policy_char)) {
            valid[index] = 0;
        }
    }
}

int main() {
    hipDeviceReset();

    std::string line;
    std::ifstream infile("inputs/day2.data");

    // Read input
    std::vector<int> copy_lower;
    std::vector<int> copy_upper;
    std::vector<char> policy;
    std::vector<std::string> passwords;
    std::regex e ("^([0-9]+)-([0-9]+) ([a-z]): ([a-z]*)$");
    std::cmatch cm;
    if (infile.is_open()) {
      while (std::getline(infile, line)) {
        std::regex_match(line.c_str(), cm, e);
        copy_lower.push_back(std::stoi(cm[1]));
        copy_upper.push_back(std::stoi(cm[2]));
        policy.push_back(cm.str(3)[0]);
        passwords.push_back(cm[4]);
      }
      infile.close();
    }

    // Restride passwords as array
    std::vector<char> passwords_strided = prepare_array(passwords);
    size_t input_len = passwords.size();

    // Copy input to device
    char* passwords_d = nullptr;
    char* policy_d = nullptr;
    int* lower_d = nullptr;
    int* upper_d = nullptr;
    array_to_device(passwords_d, passwords_strided);
    array_to_device(policy_d, policy);
    array_to_device(lower_d, copy_lower);
    array_to_device(upper_d, copy_upper);

    // Allocate space to store output (assume all valid)
    char* valid_d;
    CUDA_CALL(hipMalloc((void** )&valid_d, input_len * sizeof(char)));
    CUDA_CALL(hipMemset(valid_d, 1, input_len * sizeof(char)));

    // Check for invalid passwords
    size_t blockCount = (input_len + blockSize - 1) / blockSize;
    invalidate_passwords<<<blockCount, blockSize>>>(passwords_d,
                                                policy_d,
                                                lower_d,
                                                upper_d,
                                                valid_d,
                                                input_len);
    hipDeviceSynchronize();

    // Use device reduce sum to get the answer
    int *d_out;
    CUDA_CALL(hipMalloc((void**)&d_out, sizeof(int)));

    // Allocate temp storage
    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Sum(d_temp_storage,
                               temp_storage_bytes,
                               valid_d, d_out,
                               input_len);
    CUDA_CALL(hipMalloc(&d_temp_storage, temp_storage_bytes));

    // Sum
    hipcub::DeviceReduce::Sum(d_temp_storage,
                            temp_storage_bytes,
                            valid_d, d_out,
                            input_len);

    // Get and print answer
    int total_valid;
    CUDA_CALL(hipMemcpy(&total_valid, d_out, sizeof(int),
                        hipMemcpyDefault));
    std::cout << total_valid << std::endl;

    // Free device memory
    CUDA_CALL(hipFree(passwords_d));
    CUDA_CALL(hipFree(policy_d));
    CUDA_CALL(hipFree(lower_d));
    CUDA_CALL(hipFree(upper_d));
    CUDA_CALL(hipFree(valid_d));
    CUDA_CALL(hipFree(d_out));
    CUDA_CALL(hipFree(d_temp_storage));

    return 0;
}
