#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <vector>

#include <cub/device/device_select.cuh>

#include "../cuda.cuh"

const size_t blockSize = 32;

// Square dist matrix kernel which stores sum == 2020 in one array, multiple in another
__global__
void add_and_multiply(int* expenses_d, char* sums, int* prods, size_t length, size_t triples) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
  int count = 0;
  int i, j, k;
  // This is horrible
  if (index < triples) {
    for (i = 0; i < length - 2; ++i) {
      for (j = i + 1; j < length - 1; ++j) {
        for (k = j + 1; k < length; k++) {
          if (count == index) {
            break;
          } else {
            count++;
          }
        }
        if (count == index) {
          break;
        }
      }
      if (count == index) {
        break;
      }
    }
    __syncwarp();

    *(sums + index) = (*(expenses_d + i) + *(expenses_d + j) + *(expenses_d + k)) == 2020;
    *(prods + index) = *(expenses_d + i) * *(expenses_d + j) * *(expenses_d + k);
    //printf("idx:%d i:%d j:%d k:%d val1:%d val2:%d val3:%d sum:%d prod:%d\n",
    // index, i, j, k, *(expenses_d + i), *(expenses_d + j), *(expenses_d + k), *(sums + index), *(prods + index));
  }
}

int main() {
  std::string line;
  std::ifstream infile("inputs/day1.data");

  // Read input
  std::vector<int> expenses;
  if (infile.is_open()) {
    while (std::getline(infile, line)) {
      expenses.push_back(std::stoi(line));
    }
    infile.close();
  }

  // Copy input to device
  int* expenses_d;
  CUDA_CALL(hipMalloc((void** )&expenses_d, expenses.size() * sizeof(int)));
  CUDA_CALL(hipMemcpy(expenses_d, expenses.data(), expenses.size() * sizeof(int),
                        hipMemcpyDefault));

  // Allocate space to store output
  // nC3
  size_t n_triples = expenses.size() * (expenses.size() - 1) * (expenses.size() - 2) / 6;
  char* sums;
  CUDA_CALL(hipMalloc((void** )&sums, n_triples * sizeof(char)));
  CUDA_CALL(hipMemset(sums, 0, n_triples * sizeof(char)));
  int* prods;
  CUDA_CALL(hipMalloc((void** )&prods, n_triples * sizeof(int)));
  CUDA_CALL(hipMemset(prods, 0, n_triples * sizeof(int)));

  // Calculate sums and products
  size_t blockCount = (n_triples + blockSize - 1) / blockSize;
  add_and_multiply<<<blockCount, blockSize>>>(expenses_d,
                                              sums,
                                              prods,
                                              expenses.size(),
                                              n_triples);

  // Use device select to get the answer
  int *d_out;
  CUDA_CALL(hipMalloc((void**)&d_out, n_triples * sizeof(int)));
  int *d_num_selected_out;
  CUDA_CALL(hipMalloc((void**)&d_num_selected_out, sizeof(int)));
  void *d_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;

  // Allocate temp storage
  hipcub::DeviceSelect::Flagged(d_temp_storage,
                              temp_storage_bytes,
                              prods, sums, d_out,
                              d_num_selected_out, n_triples);
  CUDA_CALL(hipMalloc(&d_temp_storage, temp_storage_bytes));

  // Run selection
  hipcub::DeviceSelect::Flagged(d_temp_storage,
                              temp_storage_bytes,
                              prods, sums, d_out,
                              d_num_selected_out, n_triples);

  // Get and print answer
  int n_selected;
  CUDA_CALL(hipMemcpy(&n_selected, d_num_selected_out, sizeof(int),
                      hipMemcpyDefault));
  std::vector<int> answer(n_selected);
  CUDA_CALL(hipMemcpy(answer.data(), d_out, n_selected * sizeof(int),
                        hipMemcpyDefault));
  for (auto it = answer.begin(); it != answer.end(); ++it) {
    if (*it > 1) {
      std::cout << *it << std::endl;
    }
  }

  // Free device memory
  CUDA_CALL(hipFree(expenses_d));
  CUDA_CALL(hipFree(sums));
  CUDA_CALL(hipFree(prods));
  CUDA_CALL(hipFree(d_out));
  CUDA_CALL(hipFree(d_num_selected_out));
  CUDA_CALL(hipFree(d_temp_storage));

  return 0;
}